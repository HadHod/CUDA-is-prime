#include <iostream>

#include "kernel/isPrime.cu"

using namespace std;

int main(int argc, char* argv[]) {

    if (argc != 2) {
        cout << "Pass 1 integer" << endl;
        return EXIT_FAILURE;
    }

    int number = atoi(argv[1]);
    int* dev_number;

    bool* result;
    bool* dev_result;

    hipMalloc((void**) &dev_result, sizeof(bool));
    hipMalloc((void**) &dev_number, sizeof(int));

    hipMemcpy(dev_number, number, sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(number / 4, 1, 1);
    dim3 dimBlock(4, 1, 1);
    isPrime<<<dimGrid, dimBlock>>>(dev_result, number);

    hipDeviceSynchronize();

    hipMemcpy(result, dev_result, sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(dev_result);
    hipFree(dev_number);

    return EXIT_SUCCESS;
}