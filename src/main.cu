#include <iostream>

#include "kernel/main.cuh"

using namespace std;

int main(int argc, char* argv[]) {

    if (argc != 2) {
        cout << "Pass 1 integer\n";
        return EXIT_FAILURE;
    }

    int number = atoi(argv[1]);
    int* dev_number;

    bool* result;
    bool* dev_result;

    hipMalloc((void**) &dev_result, sizeof(bool));
    hipMalloc((void**) &dev_number, sizeof(int));

    hipMemcpy(dev_number, &number, sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(1, 1, 1);
    is_prime<<<dimGrid, dimBlock>>>(dev_result, number);

    hipDeviceSynchronize();

    hipMemcpy(result, dev_result, sizeof(bool), hipMemcpyDeviceToHost);

    cout << result;

    hipFree(dev_result);
    hipFree(dev_number);

    return EXIT_SUCCESS;
}
